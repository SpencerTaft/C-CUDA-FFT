#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <iostream>
#include <string>
#include <fstream>
#include <vector>
#include <complex>
#include <iostream>
#include <valarray>

/**********************************************************
 * Declarations
 **********************************************************/
//Type definitions
typedef std::complex<double> Complex;
typedef std::valarray<Complex> CArray;

template <class T> class ContiguousArray
{
public:
    unsigned int numElements;
    T* ptr;

    ContiguousArray(T* inputPtr, unsigned int inputNumElements)
    {
        ptr = inputPtr;
        numElements = inputNumElements;
    }

    ContiguousArray()
    {
        ptr = nullptr;
        numElements = 0;
    }

    unsigned int getSize()
    {
        return numElements * sizeof(T);
    }
};

typedef struct Comp
{
    float real; //real component during FFT calculation, and contains FFT magnitude after FFT calculation
    float imag;
};

//Function declarations
ContiguousArray<float> readCSV();
ContiguousArray<int> generateFrameOffsets();
ContiguousArray<float> generateWindow();
hipError_t FFTWithCuda(ContiguousArray<float> window, ContiguousArray<int> frameOffsets, ContiguousArray <float> inputArray);

//Global variables
int inputArraySize = 0;

//user set parameters
const int k_fftInputLen = 512; //length of FFT input array(data points per FFT frame)
const int k_fftFrameOffset = 100; //offset between start of FFT frames(eg x[n]=x[n-1]+k_fftFrameOffset where x[n] is the first value used as input to the fft frame)

/**********************************************************
 * Functions run on single thread
 **********************************************************/
ContiguousArray<float> readCSV()
{
    std::vector<float> csvVector;
    ContiguousArray<float> retArray;

    const char delimeter = ',';//delimeter between items in CSV file
    std::string line;
    std::string string;

    std::ifstream myFile("data.CSV");

    if (!myFile.is_open()) throw std::runtime_error("Couldn't open file");

    while (getline(myFile, string, delimeter)) {
        csvVector.push_back(std::stof(string));
        inputArraySize++;
    }

    //CUDA requires contiguous memory
    retArray.numElements = inputArraySize;
    retArray.ptr = new float[retArray.numElements];

    for (int i = 0; i < inputArraySize; i++)
    {
        retArray.ptr[i] = csvVector[i];
    }

    return retArray;
}

ContiguousArray<int> generateFrameOffsets()
{
    ContiguousArray<int> offsets;

    //CUDA requires contiguous memory
    offsets.numElements = (inputArraySize - k_fftInputLen) / k_fftFrameOffset;
    offsets.ptr = new int[offsets.numElements];

    for (int i = 0; (i * k_fftFrameOffset) <= (inputArraySize - k_fftInputLen); i++)
    {
        offsets.ptr[i] = (i * k_fftFrameOffset);
    }

    return offsets;
}

ContiguousArray<float> generateWindow()
{
    //w[n] = a0 - a1*cos(x) + a2*cos(2x) - a3cos(3x), x = (2n*pi)/N, 0 < n < N
    const float a0 = 0.35875f;
    const float a1 = 0.48829f;
    const float a2 = 0.14128f;
    const float a3 = 0.01168f;

    ContiguousArray<float> retArray;
    float x;
    float term1, term2, term3;
    float w_n;

    retArray.numElements = k_fftInputLen;
    retArray.ptr = new float[retArray.numElements];

    for (int n = 0; n < k_fftInputLen; n++)
    {
        //calculate x
        x = 2 * n * (3.14159);
        x /= k_fftInputLen;

        term1 = a1 * cos(x);
        term2 = a2 * cos(2 * x);
        term3 = a3 * cos(3 * x);

        w_n = a0 - term1 + term2 - term3;

        retArray.ptr[n] = w_n;
    }

    return retArray;
}

 /**********************************************************
  * Functions run in parallel
  **********************************************************/

__device__ void FFTkernelRecursiveCVersion(Comp* windowedDataI, int inputSize)
{
    if (inputSize <= 1)
    {
        return;
    }

    const float PI = (float)3.141592653589793238460;
    int start, size, stride;
    float polarMagnitude;
    float theta;

    //replacement for slice
    size = inputSize / 2;
    stride = 2;

    Comp* evenSlice = (Comp*)malloc(size*sizeof(Comp));
    Comp* oddSlice = (Comp*)malloc(size*sizeof(Comp));

    //divide
    for (int i = 0; i < size; i++)
    {
        evenSlice[i] = windowedDataI[(i * stride)];
        oddSlice[i] = windowedDataI[1 + (i * stride)];
    }
    
    //conquer
    FFTkernelRecursiveCVersion(evenSlice, size);
    FFTkernelRecursiveCVersion(oddSlice, size);
    
    for (size_t k = 0; k < size / 2; ++k)
    {
        
        Comp t;
        float theta = -2 * PI * k / size; //radians
        //confirmed replaces std::polar below
        t.real = (float)(cosf(theta) * oddSlice[k].real);
        t.real -= (float)(sinf(theta) * oddSlice[k].imag);

        t.imag = (float)(cosf(theta) * oddSlice[k].imag);
        t.imag += (float)(sinf(theta) * oddSlice[k].real);
        
        //x[k] = even[k] + t
        windowedDataI[k].real = evenSlice[k].real + t.real;
        windowedDataI[k].imag = evenSlice[k].imag + t.imag;

        //x[k + N / 2] = even[k] - t;
        windowedDataI[k + (size / 2)].real = evenSlice[k].real - t.real;
        windowedDataI[k + (size / 2)].imag = evenSlice[k].imag - t.imag;
    }

    free(evenSlice);
    free(oddSlice);
}

/*  Apply blackman - harris filter to input data frame.
 *  Return the result as a vector.                      */
__device__ void kernelWindowData(int frameOffset, float* windowVec, float* inputArrayVec, Comp* windowedDataI)
{
    for (int n = 0; n < k_fftInputLen; n++)
    {
        windowedDataI[n].real = windowVec[n] * inputArrayVec[n + frameOffset];
        windowedDataI[n].imag = 0.0;
    }
}

__global__ void FFTkernel(float* windowVec, int* frameOffsetsVec, float* inputArrayVec, Comp* windowedData)
{
    int i = threadIdx.x;

    //windowedData is a large array containing the memory for all fft's output data.  Each thread only writes to k_fftInputLen samples
    //This sets the pointer to the right output memory.
    int windowedDataOffset = i * k_fftInputLen;

    Comp* windowedDataI = windowedData + windowedDataOffset; 

    //Extract frame offset from frameOffsetsVec for current thread
    int frameOffset = frameOffsetsVec[i];

    //Apply windowing function to selected data
    kernelWindowData(frameOffset, windowVec, inputArrayVec, windowedDataI);

# if __CUDA_ARCH__>=200
    printf("start of FFT calc\n");
#endif
    
    //Perform FFT on windowed data, with resulting FFT written to windowedDataI
    FFTkernelRecursiveCVersion(windowedDataI, k_fftInputLen);

    //Calculate the magnitude of the real portion (up to Nyquist frequency) of the FFT, store in real component
    for (int i = 0; i < k_fftInputLen / 2; i++)
    {
        windowedDataI[i].real = windowedDataI[i].real * windowedDataI[i].real;
        windowedDataI[i].real += windowedDataI[i].imag * windowedDataI[i].imag;
        windowedDataI[i].real = sqrtf(windowedDataI[i].real);
        windowedDataI[i].imag = 0.0f; //clear imaginary component so it is clear that this value is only used during calculation
    }

# if __CUDA_ARCH__>=200
    printf("End of FFT calc\n");
#endif
}

 /**********************************************************
  * Main
  **********************************************************/
int main()
{
    std::cout << "Start of program\n";
    
    //Read CSV file and put elements in inputArray vector
    ContiguousArray<float> inputArray = readCSV();

    //generate blackman-harris filter from 0 to k_fftInputLen-1 to window the input data
    ContiguousArray<float> window = generateWindow();

    //generate frameOffsets
    ContiguousArray<int> frameOffsets = generateFrameOffsets();//list of frame offsets used by workers

    // Run FFT in parallel.
    hipError_t cudaStatus = FFTWithCuda(window, frameOffsets, inputArray);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "FFTWithCuda failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

     std::cout << "End of program\n";

    return 0;
}

hipError_t FFTWithCuda(ContiguousArray<float> window, ContiguousArray<int> frameOffsets, ContiguousArray<float> inputArray)
{
    hipError_t cudaStatus;
    std::ofstream outputFile;
    float* dev_window = 0;
    int* dev_frameOffsets = 0;
    float* dev_inputArray = 0;
    Comp* dev_windowedData = 0;
    const char delimeter = ',';

    const int threadCount = frameOffsets.numElements;
    ContiguousArray<Comp>* outputData = new ContiguousArray<Comp>[threadCount];

    // Choose which GPU to run on, change this on a multi-GPU system
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    //Allocate space on GPU for the window, frameOffsets, and inputArray.  Only one of each is needed
    cudaStatus = hipMalloc((void**)&dev_window, window.getSize());
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_frameOffsets, frameOffsets.getSize());
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_inputArray, inputArray.getSize());
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    //Allocate room on GPU for windowed data (windowing is run in parallel), data initialized on GPU so no memcpy for this data
    cudaStatus = hipMalloc((void**)&dev_windowedData, (k_fftInputLen*threadCount*sizeof(Comp)));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    //Copy input vectors from host memory to GPU buffers
    cudaStatus = hipMemcpy(dev_window, window.ptr, window.getSize(), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_frameOffsets, frameOffsets.ptr, frameOffsets.getSize(), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_inputArray, inputArray.ptr, inputArray.getSize(), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread per frameOffset
    FFTkernel << <1, threadCount >> > (dev_window, dev_frameOffsets, dev_inputArray, dev_windowedData);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "FFTKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching FFTKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory (all threads).
    for (int tIndex = 0; tIndex < threadCount; tIndex++)
    {
        outputData[tIndex].numElements = k_fftInputLen * sizeof(Comp);
        outputData[tIndex].ptr = new Comp[outputData[tIndex].numElements];

        unsigned int outputDataSize = (k_fftInputLen * sizeof(Comp));
        Comp* dev_windowedDataI = (dev_windowedData + (tIndex * k_fftInputLen));

        cudaStatus = hipMemcpy(outputData[tIndex].ptr, dev_windowedDataI, outputDataSize, hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }
    }

    //Write data to output CSV file
    outputFile.open("output.csv");

    for (int dataIndex = 0; dataIndex < k_fftInputLen/2; dataIndex++)
    {
        for (int threadIndex = 0; threadIndex < threadCount; threadIndex++)
        {
            outputFile << outputData[threadIndex].ptr[dataIndex].real;
            outputFile << delimeter;
        }
        outputFile << "\n";
    }

    outputFile.close();

Error:
    //Free input data
    hipFree(dev_window);
    hipFree(dev_frameOffsets);
    hipFree(dev_inputArray);

    //Free output data
    hipFree(dev_windowedData);

    return cudaStatus;
}

