﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>
#include <iostream>
#include <string>
#include <fstream>
#include <vector>

void readCSV();
void generateFrameOffsets();
void generateFilter();
std::vector<double> windowData(int frameOffset);
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

const char delimeter = ',';//delimeter between items in CSV file
std::vector<double>inputArray;//array of input data from CSV file
int inputArraySize = 0;
std::vector<int>frameOffsets;//list of frame offsets used by workers
std::vector<double>filter;//filter used to window the FFT frames


//user set parameters
const int k_fftInputLen = 100; //length of FFT input array
const int k_fftFrameOffset = 10; //offset between start of FFT frames(eg x[n]=x[n-1]+k_fftFrameOffset where x[n] is the first value used as input to the fft frame)

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    //>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>
    //>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>
    //>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>
    //Todo need to review garbage collection and do prototype until simple FFT is complete

    std::cout << "Start of program\n";
    readCSV();

    //generate blackman-harris filter from 0 to k_fftInputLen-1
    generateFilter();

    //generate frameOffsets
    generateFrameOffsets();

    //todo fxns below will be run in parallel

    std::vector<double> window = windowData(0);

    //<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
    //<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
    //<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    std::cout << "End of program\n";

    return 0;
}

void readCSV()
{
    std::string line;
    std::string string;

    std::ifstream myFile("data.CSV");

    if (!myFile.is_open()) throw std::runtime_error("Couldn't open file");

    while (getline(myFile, string, delimeter)) {
        //std::cout << string << std::endl;
        inputArray.push_back(std::stod(string));
        inputArraySize++;
    }
}

void generateFrameOffsets()
{
    for (int i = 0; i < (inputArraySize - k_fftInputLen); i += k_fftFrameOffset)
    {
        frameOffsets.push_back(i);
    }
}

void generateFilter()
{
    //w[n] = a0 - a1*cos(x) + a2*cos(2x) - a3cos(3x), x = (2n*pi)/N, 0 < n < N

    const double a0 = 0.35875;
    const double a1 = 0.48829;
    const double a2 = 0.14128;
    const double a3 = 0.01168;

    double x;
    double term1, term2, term3;
    double w_n;

    for (int n = 0; n < k_fftInputLen; n++)
    {
        //calculate x
        x = 2 * n * (3.14159);
        x /= k_fftInputLen;

        term1 = a1 * cos(x);
        term2 = a2 * cos(2 * x);
        term3 = a3 * cos(3 * x);

        w_n = a0 - term1 + term2 - term3;

        filter.push_back(w_n);
    }
}

/*  Apply blackman - harris filter to input data frame.
 *  Return the result as a vector.                      */
std::vector<double> windowData(int frameOffset)
{
    std::vector<double>windowedVector;
    double windowedData;

    for (int n = 0; n < k_fftInputLen; n++)
    {
        windowedData = filter[n] * inputArray[n + frameOffset];
        windowedVector.push_back(windowedData);
    }

    return windowedVector;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
