#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <iostream>
#include <string>
#include <fstream>
#include <vector>
#include <complex>
#include <iostream>
#include <valarray>

/**********************************************************
 * Declarations
 **********************************************************/
//Type definitions
typedef std::complex<double> Complex;
typedef std::valarray<Complex> CArray;

template <class T> class ContiguousArray
{
public:
    unsigned int numElements;
    T* ptr;

    ContiguousArray(T* inputPtr, unsigned int inputNumElements)
    {
        ptr = inputPtr;
        numElements = inputNumElements;
    }

    ContiguousArray()
    {
        ptr = nullptr;
        numElements = 0;
    }

    unsigned int getSize()
    {
        return numElements * sizeof(T);
    }
};

//Function declarations
ContiguousArray<double> readCSV();
std::vector<int> generateFrameOffsets();
ContiguousArray<double> generateFilter();
std::vector<double> windowData(int frameOffset, std::vector<double> filter);
void fft(CArray& x);
hipError_t FFTWithCuda(ContiguousArray<double> filter, const std::vector<int>& frameOffsets, ContiguousArray<double> inputArray);

//Global variables
int inputArraySize = 0;

//Global FFT variables
const double PI = 3.141592653589793238460;
typedef std::complex<double> Complex;
typedef std::valarray<Complex> CArray;

//user set parameters
const int k_fftInputLen = 100; //length of FFT input array(data points per FFT frame)
const int k_fftFrameOffset = 10; //offset between start of FFT frames(eg x[n]=x[n-1]+k_fftFrameOffset where x[n] is the first value used as input to the fft frame)

/**********************************************************
 * Functions run on single thread
 **********************************************************/
ContiguousArray<double> readCSV()
{
    std::vector<double> csvVector;
    ContiguousArray<double> retArray;

    const char delimeter = ',';//delimeter between items in CSV file
    std::string line;
    std::string string;

    std::ifstream myFile("data.CSV");

    if (!myFile.is_open()) throw std::runtime_error("Couldn't open file");

    while (getline(myFile, string, delimeter)) {
        csvVector.push_back(std::stod(string));
        inputArraySize++;
    }

    //CUDA requires contiguous memory
    retArray.ptr = new double[inputArraySize];
    retArray.numElements = inputArraySize;

    for (int i = 0; i < inputArraySize; i++)
    {
        retArray.ptr[i] = csvVector[i];
    }

    return retArray;
}

std::vector<int> generateFrameOffsets()
{
    std::vector<int> offsets;

    for (int i = 0; i < (inputArraySize - k_fftInputLen); i += k_fftFrameOffset)
    {
        offsets.push_back(i);
    }

    return offsets;
}

ContiguousArray<double> generateFilter()
{
    //w[n] = a0 - a1*cos(x) + a2*cos(2x) - a3cos(3x), x = (2n*pi)/N, 0 < n < N
    const double a0 = 0.35875;
    const double a1 = 0.48829;
    const double a2 = 0.14128;
    const double a3 = 0.01168;

    ContiguousArray<double> retArray;
    double x;
    double term1, term2, term3;
    double w_n;

    retArray.ptr = new double[k_fftInputLen];
    retArray.numElements = k_fftInputLen;

    for (int n = 0; n < k_fftInputLen; n++)
    {
        //calculate x
        x = 2 * n * (3.14159);
        x /= k_fftInputLen;

        term1 = a1 * cos(x);
        term2 = a2 * cos(2 * x);
        term3 = a3 * cos(3 * x);

        w_n = a0 - term1 + term2 - term3;

        retArray.ptr[n] = w_n;
    }

    return retArray;
}

 /**********************************************************
  * Functions run in parallel
  **********************************************************/

// Cooley–Tukey FFT (in-place)
void fft(CArray& x)
{
    const size_t N = x.size();
    if (N <= 1) return;

    // divide
    CArray even = x[std::slice(0, N / 2, 2)];
    CArray  odd = x[std::slice(1, N / 2, 2)];

    // conquer
    fft(even);
    fft(odd);

    // combine
    for (size_t k = 0; k < N / 2; ++k)
    {
        Complex t = std::polar(1.0, -2 * PI * k / N) * odd[k];
        x[k] = even[k] + t;         //todo this is overwriting the input data, need to preserve input copy
        x[k + N / 2] = even[k] - t; //todo this is overwriting the input data, need to preserve input copy
    }
}

/*  Apply blackman - harris filter to input data frame.
 *  Return the result as a vector.                      */
std::vector<double> windowData(int frameOffset, std::vector<double> filter, std::vector<double> inputArray)
{
    std::vector<double>windowedVector;
    double windowedData;

    for (int n = 0; n < k_fftInputLen; n++)
    {
        windowedData = filter[n] * inputArray[n + frameOffset];
        windowedVector.push_back(windowedData);
    }

    return windowedVector;
}

__device__ void kernelWindowData(int frameOffset, double* filterVec, double* inputArrayVec, double* windowedDataI)
{
    for (int n = 0; n < k_fftInputLen; n++)
    {
        windowedDataI[n] = filterVec[n] * inputArrayVec[n + frameOffset];

        # if __CUDA_ARCH__>=200
        //printf("%f \n", windowedDataI[n]);
        printf("%f \n", filterVec[n]);
        //printf("%f \n", inputArrayVec[n]);
        #endif
    }
}

__device__ void FFTkernelRecursive(int i)
{
    //do nothing
    //fft(data);
}

//todo this needs to receive pointer to return memory
__global__ void FFTkernel(double* filterVec, int* frameOffsetsVec, double* inputArrayVec, double* windowedData)
{
    //Todo skip window for now, add once I get the raw FFT working on GPU
    //std::vector<double> windowedData = windowData(0, filter, inputArray);

    //windowData would return a vector with length k_fftInputLen and applies the frame offset.  for testing, run FFT on the first frame

    int i = threadIdx.x;

    int windowedDataOffset = i * k_fftInputLen;
    double* windowedDataI = windowedData + windowedDataOffset;

    # if __CUDA_ARCH__>=200
    //printf("%f \n", windowedDataI);
    #endif  

    kernelWindowData(i, filterVec, inputArrayVec, windowedDataI);
    FFTkernelRecursive(i);
}

 /**********************************************************
  * Main
  **********************************************************/
int main()
{
    std::cout << "Start of program\n";

    //Read CSV file and put elements in inputArray vector
    ContiguousArray<double> inputArray = readCSV();

    //generate blackman-harris filter from 0 to k_fftInputLen-1 to window the input data
    ContiguousArray<double> filter = generateFilter();

    //generate frameOffsets
    const std::vector<int> frameOffsets = generateFrameOffsets();//list of frame offsets used by workers

    // Run FFT in parallel.
    hipError_t cudaStatus = FFTWithCuda(filter, frameOffsets, inputArray);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "FFTWithCuda failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    std::cout << "End of program\n";

    return 0;
}

hipError_t FFTWithCuda(ContiguousArray<double> filter, const std::vector<int>& frameOffsets, ContiguousArray<double> inputArray)
{
    hipError_t cudaStatus;
    double* dev_filter = 0;
    int* dev_frameOffsets = 0;
    double* dev_inputArray = 0;

    //todo debug, only run one thread until that case works
    const int const threadCount = 1;///////////////////////////frameOffsets.size();
    std::vector<double> emptyWindowedData;

    double* dev_windowedData = 0;

    // Choose which GPU to run on, change this on a multi-GPU system
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    //Allocate space on GPU for the filter, frameOffsets, and inputArray.  Only one of each is needed

    cudaStatus = hipMalloc((void**)&dev_filter, filter.getSize());
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_frameOffsets, sizeof(frameOffsets) * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_inputArray, inputArray.getSize());
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    //Allocate room on GPU for windowed data (windowing is run in parallel), data initialized on GPU so no memcpy for this data

    for (int i = 0; i < (k_fftInputLen*threadCount); i++)
    {
        emptyWindowedData.push_back(0.0);
    }

    cudaStatus = hipMalloc((void**)&dev_windowedData, (k_fftInputLen*threadCount));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_windowedData, &emptyWindowedData[0], sizeof(emptyWindowedData), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    //Copy input vectors from host memory to GPU buffers
    
    cudaStatus = hipMemcpy(dev_filter, filter.ptr, filter.getSize(), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_frameOffsets, &frameOffsets[0], sizeof(frameOffsets)*sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_inputArray, inputArray.ptr, inputArray.getSize(), hipMemcpyHostToDevice);//todo replace global
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread per frameOffset

    FFTkernel << <1, threadCount >> > (dev_filter, dev_frameOffsets, dev_inputArray, dev_windowedData);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    //emptyWindowedData now contains the windowedData
    cudaStatus = hipMemcpy(&emptyWindowedData[0], dev_windowedData, sizeof(emptyWindowedData), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    //Free input data
    hipFree(dev_filter);
    hipFree(dev_frameOffsets);
    hipFree(dev_inputArray);

    //Free output data
    hipFree(dev_windowedData);

    return cudaStatus;
}